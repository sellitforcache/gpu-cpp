#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"
#include "wfloat3.h"


__global__ void escatter_kernel(unsigned N, unsigned RNUM_PER_THREAD, unsigned* isonum, unsigned * index, float * rn_bank, float * E, source_point * space, unsigned * rxn, float * awr_list, unsigned* done, float** scatterdat){


	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid >= N){return;}       //return if out of bounds
	if (done[tid]){return;}
	if (rxn[tid] != 2){return;}  //return if not elastic scatter

	//printf("in escatter\n");

	//constants
	const float  pi           =   3.14159265359 ;
	const float  m_n          =   1.00866491600 ; // u
	const float  temp         =   0.02585202857e-6;    // MeV
	const float  E_cutoff     =   1e-11;
	const float  E_max        =   20.0; //MeV
	// load history data
	unsigned 	this_tope 	= isonum[tid];
	unsigned 	this_dex	= index[tid];  //this is no longer the row, it is now the exact index of the array, set by microscopic
	float 		this_E 		= E[tid];
	//float 		this_Q 		= 0.0;
	wfloat3 	hats_old(space[tid].xhat,space[tid].yhat,space[tid].zhat);
	float 		this_awr	= awr_list[this_tope];
	float * 	this_array 	= scatterdat[this_dex];
	float 		rn1 		= rn_bank[ tid*RNUM_PER_THREAD + 3];
	float 		rn2 		= rn_bank[ tid*RNUM_PER_THREAD + 4];
	float 		rn3 		= rn_bank[ tid*RNUM_PER_THREAD + 5];
	float 		rn4 		= rn_bank[ tid*RNUM_PER_THREAD + 6];
	float 		rn5 		= rn_bank[ tid*RNUM_PER_THREAD + 7];
	float 		rn6 		= rn_bank[ tid*RNUM_PER_THREAD + 8];
	float 		rn7 		= rn_bank[ tid*RNUM_PER_THREAD + 9];
	float 		rn8 		= rn_bank[ tid*RNUM_PER_THREAD + 10];
	//float 		rn9 		= rn_bank[ tid*RNUM_PER_THREAD + 11];

	// internal kernel variables
	float 		mu, phi, next_E, last_E;
    unsigned 	vlen, next_vlen, offset; 
    unsigned  	isdone = 0;
	float  		E_target     		=   temp * ( -logf(rn1) - logf(rn2)*cosf(pi/2*rn3)*cosf(pi/2*rn3) );
	float 		speed_target     	=   sqrtf(2.0*E_target/(this_awr*m_n));
	float  		speed_n          	=   sqrtf(2.0*this_E/m_n);
	float 		E_new				=   0.0;
	//float 		a 					= 	this_awr/(this_awr+1.0);
	wfloat3 	v_n_cm,v_t_cm,v_n_lf,v_t_lf,v_cm, hats_new, hats_target;
	float 		mu0,mu1,cdf0,cdf1;
	//float 		v_rel,E_rel;

	// make target isotropic
	mu  = (2.0*rn4) - 1.0;
	phi = 2.0*pi*rn5;
	hats_target.x = sqrtf(1.0-(mu*mu))*cosf(phi);
	hats_target.y = sqrtf(1.0-(mu*mu))*sinf(phi); 
	hats_target.z = mu;
	
	// make speed vectors
	v_n_lf = hats_old    * speed_n;
	v_t_lf = hats_target * speed_target;

	// calculate  v_cm
	v_cm = (v_n_lf + (v_t_lf*this_awr))/(1.0+this_awr);

	//transform neutron velocity into CM frame
	v_n_cm = v_n_lf - v_cm;
	v_t_cm = v_t_lf - v_cm;
	//printf("cm=(% 6.4E % 6.4E % 6.4E)\n",v_n_cm.x,v_n_cm.y,v_n_cm.z);
	//wfloat3 crossit = v_n_cm.cross(v_t_cm);
	//printf("crossmag=% 6.4E\n",crossit.norm2());
	

	// sample new phi, mu_cm
	phi = 2.0*pi*rn7;
	offset=4;
	if(this_array == 0x0){
		mu= 2*rn6-1; //MT=91 doesn't have angular tables for whatever reason
	}
	else{  // 
		memcpy(&last_E, 	&this_array[0], sizeof(float));
		memcpy(&next_E, 	&this_array[1], sizeof(float));
		memcpy(&vlen, 		&this_array[2], sizeof(float));
		memcpy(&next_vlen, 	&this_array[3], sizeof(float));
		//printf("(last,this,next) = %6.4E %6.4E %6.4E, prob=%6.4E, (this,next)_vlen= %u %u\n",last_E,this_E,next_E,(next_E-this_E)/(next_E-last_E),vlen,next_vlen);
		if(  rn8 <= (next_E-this_E)/(next_E-last_E) ){   //sample last E
			for(unsigned k=0;k<vlen-1;k++){
				if(rn6 <= this_array[offset+vlen+(k+1)] ){  //look at CDF one ahead sicne first is 0
					//in this bin, linearly interpolate 
					mu0 	= this_array[offset       + k  ];
					mu1  	= this_array[offset       + k+1];
					cdf0 	= this_array[offset+vlen  + k  ];
					cdf1 	= this_array[offset+vlen  + k+1];
					mu 		= (mu1-mu0)/(cdf1-cdf0)*(rn6-cdf0)+mu0;
					break;
				}
			}
		}
		else{   // sample E+1
			for(unsigned k=0;k<next_vlen-1;k++){
				if(rn6 <= this_array[offset+(2*vlen)+next_vlen+(k+1)] ){  //look at CDF one ahead sicne first is 0
					//in this bin, linearly interpolate 
					mu0 	= this_array[offset+(2*vlen)            + k  ];
					mu1  	= this_array[offset+(2*vlen)            + k+1];
					cdf0 	= this_array[offset+(2*vlen)+next_vlen  + k  ];
					cdf1 	= this_array[offset+(2*vlen)+next_vlen  + k+1];
					mu 		= (mu1-mu0)/(cdf1-cdf0)*(rn6-cdf0)+mu0;
					break;
				}
			}
		}
		//if(this_E >= 1.03  & this_E < 1.04){
		//	printf("%d %10.8E %u %10.8E %10.8E\n",tid,this_E,vlen,rn6,mu);
		//}
	}


	// pre rotation directions
	hats_old = v_n_cm / v_n_cm.norm2();
	//  create a perpendicular roation vector 
	//wfloat3 rotation_hat( 0.0, 0.0, 1.0 );
	wfloat3 rotation_hat = hats_target.cross( v_n_cm );
	rotation_hat = rotation_hat / rotation_hat.norm2();
	//  do rotations, polar first, then azimuthal
	v_n_cm.rodrigues_rotation( rotation_hat, acosf(mu) );
	v_n_cm.rodrigues_rotation( hats_old,     phi       );
	// transform back to L
	v_n_lf = v_n_cm + v_cm;
	hats_new = v_n_lf / v_n_lf.norm2();
	// calculate energy
	E_new = 0.5 * m_n * v_n_lf.dot(v_n_lf);

	//if (this_E>=0.4){
	//	E_new = this_E*(     (1.0 + this_awr*this_awr + 2.0*this_awr*mu) / ( (1.0+this_awr)*(1.0+this_awr) )      );
	//}

	// enforce limits
	if ( E_new <= E_cutoff | E_new > E_max ){
		isdone=1;
	}

	//printf("speed target = %6.4E, speed=%6.4E, Eold,Enew = %10.8E %10.8E\n",speed_target, speed_n,this_E,E_new);
	// write results
	done[tid]       = isdone;
	E[tid]          = E_new;
	space[tid].xhat = hats_new.x;
	space[tid].yhat = hats_new.y;
	space[tid].zhat = hats_new.z;


}

void escatter(unsigned blks, unsigned NUM_THREADS, unsigned N, unsigned RNUM_PER_THREAD, unsigned* isonum, unsigned * index, float * rn_bank, float * E, source_point * space ,unsigned * rxn, float* awr_list, unsigned* done, float** scatterdat){

	escatter_kernel <<< blks, NUM_THREADS >>> (  N, RNUM_PER_THREAD, isonum, index, rn_bank, E, space, rxn, awr_list, done, scatterdat);
	hipDeviceSynchronize();

}

