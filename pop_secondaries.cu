#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"
#include "LCRNG.cuh"

__global__ void pop_secondaries_kernel(unsigned N, unsigned RNUM_PER_THREAD, unsigned* completed, unsigned* scanned, unsigned* yield, unsigned* done, unsigned* index, unsigned* rxn, source_point* space, float* E , unsigned* rn_bank, float**  energydata){

	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid >= N){return;}
	if (yield[tid]==0){return;}
	//if(done[tid]){return;}

	// external data
	unsigned 		position 	= scanned[tid];
	unsigned 		this_yield 	= yield[tid];
	unsigned 		dex  		= index[tid];
	float 			this_E 		= E[tid];
	//unsigned 		this_rxn 	= rxn[tid];
	float * 		this_array 	= energydata[dex];
	unsigned 		data_dex 	= 0;
	source_point 	this_space 	= space[tid];
	unsigned 		rn 			= rn_bank[tid];

	// internal data
	float 		Emin=1e-11;
	float 		Emax=20.0;
	unsigned 	k, n, offset, vlen, next_vlen, law;
	float 		sampled_E, phi, mu, rn1, rn2, last_E, next_E, e_start, E0, E1, Ek, next_e_start, next_e_end, last_e_start, last_e_end, diff;;
	float 		cdf0, cdf1, e0, e1, m, pdf0, pdf1, arg;
	const float  pi           =   3.14159265359 ;

	// sample spectrum, set data.  
	// reset self then write elsewhere

	//read in values
	rn1 = get_rand(&rn);
	rn2 = get_rand(&rn);
	offset = 5;
	//printf("rxn %u eptr %p\n",this_rxn,this_array);
	memcpy(&last_E,   	&this_array[0], sizeof(float));
	memcpy(&next_E,   	&this_array[1], sizeof(float));
	memcpy(&vlen,   	&this_array[2], sizeof(float));
	memcpy(&next_vlen,	&this_array[3], sizeof(float));
	memcpy(&law, 		&this_array[4], sizeof(float));
	float r = (this_E-last_E)/(next_E-last_E);
	last_e_start = this_array[ offset ];
	last_e_end   = this_array[ offset + vlen - 1 ];
	next_e_start = this_array[ offset + 3*vlen ];
	next_e_end   = this_array[ offset + 3*vlen + next_vlen - 1];
	//printf("rxn=%u law=%u vlen/next= %u %u, E-last/this/next= %6.4E %6.4E %6.4E\n",this_rxn,law,vlen,next_vlen,last_E,this_E,next_E);
	//sample energy dist
	sampled_E = 0.0;
	if(  rn2 >= r ){   //sample last E
		diff = next_e_end - next_e_start;
		e_start = next_e_start;
		for ( n=0 ; n<vlen-1 ; n++ ){
			cdf0 		= this_array[ (offset +   vlen ) + n+0];
			cdf1 		= this_array[ (offset +   vlen ) + n+1];
			pdf0		= this_array[ (offset + 2*vlen ) + n+0];
			pdf1		= this_array[ (offset + 2*vlen ) + n+1];
			e0  		= this_array[ (offset          ) + n+0];
			e1  		= this_array[ (offset          ) + n+1]; 
			if( rn1 >= cdf0 & rn1 < cdf1 ){
				break;
			}
		}
	}
	else{
		diff = next_e_end - next_e_start;
		e_start = next_e_start;
		for ( n=0 ; n<next_vlen-1 ; n++ ){
			cdf0 		= this_array[ (offset + 3*vlen +   next_vlen ) + n+0];
			cdf1  		= this_array[ (offset + 3*vlen +   next_vlen ) + n+1];
			pdf0		= this_array[ (offset + 3*vlen + 2*next_vlen ) + n+0];
			pdf1		= this_array[ (offset + 3*vlen + 2*next_vlen ) + n+1];
			e0   		= this_array[ (offset + 3*vlen               ) + n+0];
			e1   		= this_array[ (offset + 3*vlen               ) + n+1];
			if( rn1 >= cdf0 & rn1 < cdf1 ){
				break;
			}
		}
	}

	// interpolate the values
	m 			= (pdf1 - pdf0)/(e1-e0);
	arg = pdf0*pdf0 + 2.0 * m * (rn1-cdf0);
	if(arg<0){arg=0.0;}
	E0 	= e0 + (  sqrtf( arg ) - pdf0) / m ;
	//sampled_E = e0 + (rn1-cdf0)/pdf0;
	//printf("%u %u %u %u %u %p %6.4E %u %u %6.4E %6.4E %6.4E %6.4E %6.4E %6.4E %6.4E %6.4E %6.4E\n",tid,tid*RNUM_PER_THREAD + 12,fork,n,dex,this_array,rn1,next_vlen,vlen,this_E,e0,e1,cdf0,cdf1,pdf0,pdf1,m,sampled_E);

	// scale it
	E1 = last_e_start + r*( next_e_start - last_e_start );
	Ek = last_e_end   + r*( next_e_end   - last_e_end   );
	sampled_E = E1 +(E0-e_start)*(Ek-E1)/diff;

	//sample isotropic directions
	rn1 = get_rand(&rn);
	rn2 = get_rand(&rn);
	mu  = 2.0*rn1-1.0; 
	phi = 2.0*pi*rn2;
	
	//check limits
	if (sampled_E >= Emax){sampled_E = Emax * 0.9;}//printf("enforcing limits in pop data_dex=%u, sampled_E = %6.4E\n",data_dex,sampled_E);}
	if (sampled_E <= Emin){sampled_E = Emin * 1.1;}//printf("enforcing limits in pop data_dex=%u, sampled_E = %6.4E\n",data_dex,sampled_E);}

	// sync before writes
	__syncthreads();

	// set self data
	E    [ tid ] 		= sampled_E;
	space[ tid ].xhat 	= sqrtf(1.0-(mu*mu))*cosf(phi);
	space[ tid ].yhat 	= sqrtf(1.0-(mu*mu))*sinf(phi); 
	space[ tid ].zhat 	= mu;
	done [ tid ] 		= 0;
	yield[ tid ] 		= 0;
	rxn  [ tid ] 		= 0;//this_rxn;
	//printf("popped - dex %u rxn %u ptr %p sampled_E %6.4E\n",tid,this_rxn,this_array,sampled_E); 

	for(k=0 ; k < this_yield-1 ; k++ ){
		//get proper data index
		data_dex=completed[position+k];
		//printf("tid %u position %u k %u data_dex %u done %u (xyz) % 6.4E % 6.4E % 6.4E\n",tid,position,k,data_dex,done[data_dex],this_space.x,this_space.y,this_space.z);
		//make sure data is done
		if(!done[data_dex]){printf("overwriting into active data!\n");}
		//copy in values
		rn1 = get_rand(&rn);
		rn2 = get_rand(&rn);
		//rn1 = rn_bank[ tid*RNUM_PER_THREAD + 11 + (k+1)*4];
		//rn2 = rn_bank[ tid*RNUM_PER_THREAD + 12 + (k+1)*4];
		//sample energy dist
		sampled_E = 0.0;
		if(  rn2 >= r ){   //sample last E
			diff = next_e_end - next_e_start;
			e_start = next_e_start;
			for ( n=0 ; n<vlen-1 ; n++ ){
				cdf0 		= this_array[ (offset +   vlen ) + n+0];
				cdf1 		= this_array[ (offset +   vlen ) + n+1];
				pdf0		= this_array[ (offset + 2*vlen ) + n+0];
				pdf1		= this_array[ (offset + 2*vlen ) + n+1];
				e0  		= this_array[ (offset          ) + n+0];
				e1  		= this_array[ (offset          ) + n+1]; 
				if( rn1 >= cdf0 & rn1 < cdf1 ){
					break;
				}
			}
		}
		else{
			diff = next_e_end - next_e_start;
			e_start = next_e_start;
			for ( n=0 ; n<next_vlen-1 ; n++ ){
				cdf0 		= this_array[ (offset + 3*vlen +   next_vlen ) + n+0];
				cdf1  		= this_array[ (offset + 3*vlen +   next_vlen ) + n+1];
				pdf0		= this_array[ (offset + 3*vlen + 2*next_vlen ) + n+0];
				pdf1		= this_array[ (offset + 3*vlen + 2*next_vlen ) + n+1];
				e0   		= this_array[ (offset + 3*vlen               ) + n+0];
				e1   		= this_array[ (offset + 3*vlen               ) + n+1];
				if( rn1 >= cdf0 & rn1 < cdf1 ){
					break;
				}
			}
		}
	
		// interpolate the values
		m   = (pdf1 - pdf0)/(e1-e0);
		arg = pdf0*pdf0 + 2.0 * m * (rn1-cdf0);
		if(arg<0){arg=0.0;}
		E0 	= e0 + (  sqrtf( arg ) - pdf0) / m ;
		//sampled_E = e0 + (rn1-cdf0)/pdf0;
		//printf("%u %u %u %u %u %p %6.4E %u %u %6.4E %6.4E %6.4E %6.4E %6.4E %6.4E %6.4E %6.4E %6.4E\n",tid,tid*RNUM_PER_THREAD + 11 + (k+1)*3,fork,n,dex,this_array,rn1,next_vlen,vlen,this_E,e0,e1,cdf0,cdf1,pdf0,pdf1,m,sampled_E);

		// scale it
		E1 = last_e_start + r*( next_e_start - last_e_start );
		Ek = last_e_end   + r*( next_e_end   - last_e_end   );
		sampled_E = E1 +(E0-e_start)*(Ek-E1)/diff;

		//sample isotropic directions
		rn1 = get_rand(&rn);
		rn2 = get_rand(&rn);
		mu  = 2.0*rn1-1.0; 
		phi = 2.0*pi*rn2;
	
		//printf("tid %u k %u mu % 6.4E phi % 6.4E rn1 % 6.4E rn2 % 6.4E compactpos %u realpos %u\n",tid,k,mu,phi,rn1,rn2, position, completed[k+position]);
	
		//check data
		//printf("done? %u\n",done[ data_dex ]);
	
		//check limits
		if (sampled_E >= Emax){sampled_E = Emax * 0.9;}//printf("enforcing limits in pop data_dex=%u, sampled_E = %6.4E\n",data_dex,sampled_E);}
		if (sampled_E <= Emin){sampled_E = Emin * 1.1;}//printf("enforcing limits in pop data_dex=%u, sampled_E = %6.4E\n",data_dex,sampled_E);}


		// sync before writes
		__syncthreads();

		// set data
		E    [ data_dex ] 		= sampled_E;
		space[ data_dex ].x 	= this_space.x;
		space[ data_dex ].y 	= this_space.y;
		space[ data_dex ].z 	= this_space.z;
		space[ data_dex ].xhat 	= sqrtf(1.0-(mu*mu))*cosf(phi);
		space[ data_dex ].yhat 	= sqrtf(1.0-(mu*mu))*sinf(phi); 
		space[ data_dex ].zhat 	= mu;
		done [ data_dex ] 		= 0;
		yield[ data_dex ] 		= 0;
		rxn  [ data_dex ]		= 0;//this_rxn;
		//printf("popped - dex %u rxn %u ptr %p sampled_E %6.4E\n",data_dex,this_rxn,this_array,sampled_E); 

	}

	rn_bank[tid] = rn;

}

void pop_secondaries( unsigned NUM_THREADS,  unsigned N, unsigned RNUM_PER_THREAD, unsigned* d_completed, unsigned* d_scanned, unsigned* d_yield, unsigned* d_done, unsigned* d_index, unsigned* d_rxn, source_point* d_space, float* d_E , unsigned* d_rn_bank, float ** energydata){

	unsigned blks = ( N + NUM_THREADS - 1 ) / NUM_THREADS;

	pop_secondaries_kernel <<< blks, NUM_THREADS >>> ( N, RNUM_PER_THREAD, d_completed, d_scanned, d_yield, d_done, d_index, d_rxn, d_space, d_E , d_rn_bank, energydata);
	hipDeviceSynchronize();

}

