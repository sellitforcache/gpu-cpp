#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"

__global__ void absorb_kernel(unsigned N, unsigned * rxn , unsigned* done){


	//PLACEHOLDER FOR FISSIONS, NEED TO READ NU TABLES LATER
	
	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid >= N){return;}       //return if out of bounds
	if (done[tid]){return;}      // return if done, duh
	if (rxn[tid] >= 102 & rxn[tid] <= 113 ){return;}  //return if no secondary neutron

	done[tid]  = 1;

}

void absorb(unsigned blks, unsigned NUM_THREADS, unsigned N, unsigned * rxn , unsigned* done){

	absorb_kernel <<< blks, NUM_THREADS >>> (  N,  rxn , done);
	hipDeviceSynchronize();

}

