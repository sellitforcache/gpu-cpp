#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "datadef.h"

using namespace optix;

rtBuffer<geom_data,1>               dims;
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(unsigned,  cellnum,     attribute cell_num, );
rtDeclareVariable(unsigned,  cellmat,     attribute cell_mat, );
rtDeclareVariable(unsigned,  cellfissile, attribute cell_fis, );

RT_PROGRAM void intersect(int object_dex)
{
  float3 mins = make_float3(dims[object_dex].min[0],dims[object_dex].min[1],dims[object_dex].min[2]);
  float3 maxs = make_float3(dims[object_dex].max[0],dims[object_dex].max[1],dims[object_dex].max[2]);
  float3 loc  = make_float3(dims[object_dex].loc[0],dims[object_dex].loc[1],dims[object_dex].loc[2]);
  float3 xformed_origin = ray.origin - loc;

  //rtPrintf("%6.4E %6.4E %6.4E %6.4E %6.4E %6.4E %6.4E %6.4E %6.4E\n",xformed_origin.x,xformed_origin.y,xformed_origin.z,mins.x,mins.y,mins.z,maxs.x,maxs.y,maxs.z);

  float3 t0 = (mins - xformed_origin)/ray.direction;
  float3 t1 = (maxs - xformed_origin)/ray.direction;
  float3 near = fminf(t0, t1);
  float3 far  = fmaxf(t0, t1);
  float tmin  = fmaxf( near );
  float tmax  = fminf( far );

  //rtPrintf("%6.4E %6.4E\n",tmin,tmax);

  if(tmin <= tmax) {
    bool check_second = true;
    if( rtPotentialIntersection( tmin ) ) {
        cellnum     = dims[object_dex].cellnum;
        cellmat     = dims[object_dex].matnum;
        cellfissile = dims[object_dex].is_fissile;
       if(rtReportIntersection(0))
         check_second = false;
    } 
    if(check_second) {
      if( rtPotentialIntersection( tmax ) ) {
         cellnum     = dims[object_dex].cellnum;
         cellmat     = dims[object_dex].matnum;
         cellfissile = dims[object_dex].is_fissile;
        rtReportIntersection(0);
      }
    }
  }
}

RT_PROGRAM void bounds (int object_dex, float result[6])
{
  float3 mins = make_float3(dims[object_dex].min[0],dims[object_dex].min[1],dims[object_dex].min[2]);
  float3 maxs = make_float3(dims[object_dex].max[0],dims[object_dex].max[1],dims[object_dex].max[2]);
  float3 loc  = make_float3(dims[object_dex].loc[0],dims[object_dex].loc[1],dims[object_dex].loc[2]);

  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(mins+loc, maxs+loc);
}
