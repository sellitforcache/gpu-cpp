#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "datadef.h"

using namespace optix;

rtBuffer<geom_data,1>               dims;
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(unsigned,  cellnum,     attribute cell_num, );
rtDeclareVariable(unsigned,  cellmat,     attribute cell_mat, );
rtDeclareVariable(unsigned,  cellfissile, attribute cell_fis, );

RT_PROGRAM void intersect(int object_dex)
{
  float3 mins = make_float3(dims[object_dex].min[0],dims[object_dex].min[1],dims[object_dex].min[2]);
  float3 maxs = make_float3(dims[object_dex].max[0],dims[object_dex].max[1],dims[object_dex].max[2]);
  float3 t0 = (mins - ray.origin)/ray.direction;
  float3 t1 = (maxs - ray.origin)/ray.direction;
  float3 near = fminf(t0, t1);
  float3 far = fmaxf(t0, t1);
  float tmin = fmaxf( near );
  float tmax = fminf( far );

  if(tmin <= tmax) {
    bool check_second = true;
    if( rtPotentialIntersection( tmin ) ) {
        cellnum     = dims[object_dex].cellnum;
        cellmat     = dims[object_dex].matnum;
        cellfissile = dims[object_dex].is_fissile;
       if(rtReportIntersection(0))
         check_second = false;
    } 
    if(check_second) {
      if( rtPotentialIntersection( tmax ) ) {
        cellnum     = dims[object_dex].cellnum;
         cellmat     = dims[object_dex].matnum;
         cellfissile = dims[object_dex].is_fissile;
        rtReportIntersection(0);
      }
    }
  }
}

RT_PROGRAM void bounds (int object_dex, float result[6])
{
  float3 mins = make_float3(dims[object_dex].min[0],dims[object_dex].min[1],dims[object_dex].min[2]);
  float3 maxs = make_float3(dims[object_dex].max[0],dims[object_dex].max[1],dims[object_dex].max[2]);
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(mins, maxs);
}
