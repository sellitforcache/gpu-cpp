#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"

__global__ void find_E_grid_index_kernel(unsigned N, unsigned N_energies, unsigned* active, float * main_E_grid, float* E , unsigned * index, unsigned* done){

	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid >= N){return;}
	
	// remap to active
	tid=active[tid];

	// load data
	float value = E[tid];
	unsigned donesearching = 0;
	unsigned cnt  = 1;
	unsigned powtwo = 2;
	int dex  = (N_energies-1) / 2;  //N_energiesgth starts at 1, duh

	//printf("%p %d %10.4E\n",main_E_grid,dex,value);
	//int k;

	while(!donesearching){
		powtwo = powtwo * 2;
		if      ( 	main_E_grid[dex]   <= value && 
					main_E_grid[dex+1] >  value ) { donesearching = 1; }
		else if ( 	main_E_grid[dex]   >  value ) { dex  = dex - ((N_energies / powtwo) + 1) ; cnt++; }  // +1's are to do a ceiling instead of a floor on integer division
		else if ( 	main_E_grid[dex]   <  value ) { dex  = dex + ((N_energies / powtwo) + 1) ; cnt++; }

		if(cnt>30){
			donesearching=1;
			printf("binary search iteration overflow! %p %d % 10.8f tid=%u\n",main_E_grid,N_energies,value,tid);
			dex=0;
		}

		// edge checks... fix later???
		if(dex<0){
			//printf("binary search error! dex=%d, (ptr,N_energies,value) %p %d % 10.8f\n",dex,main_E_grid,N_energies,value);
			//for(k=0;k<N_energies;k++){printf("%10.8E\n",main_E_grid[k]);}
			dex=0;
			//donesearching=1;
		}
		if(dex>=N_energies){
			//printf("binary search error! dex=%d, (ptr,N_energies,value) %p %d % 10.8f\n",dex,main_E_grid,N_energies,value);
			//for(k=0;k<N_energies;k++){printf("%10.8E\n",main_E_grid[k]);}
			dex=N_energies-1;
			//donesearching=1;
		}
	}


	//write output index
	index[tid]=dex;

}

void find_E_grid_index(unsigned NUM_THREADS, unsigned N, unsigned N_energies,unsigned* active, float * main_E_grid, float* E , unsigned * index , unsigned* done){

	unsigned blks = ( N + NUM_THREADS - 1 ) / NUM_THREADS;

	find_E_grid_index_kernel <<< blks, NUM_THREADS >>> ( N, N_energies, active, main_E_grid,  E , index , done);
	hipDeviceSynchronize();

}

