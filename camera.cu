#include "hip/hip_runtime.h"
#include "optix.h"
#include <optix_world.h>
#include "datadef.h"

using namespace optix;

rtBuffer<source_point,1>            positions_buffer;
rtBuffer<unsigned,1>                rxn_buffer;
rtBuffer<unsigned,1>                done_buffer;
rtBuffer<unsigned,1>                cellnum_buffer;
rtBuffer<unsigned,1>                matnum_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim,   rtLaunchDim, );
rtDeclareVariable(unsigned,  outer_cell, , );
rtDeclareVariable(unsigned,  trace_type, , );
rtDeclareVariable(unsigned,  boundary_condition, , );

RT_PROGRAM void camera()
{

  if(done_buffer[launch_index]){return;}

  // declare important stuff
  int                 cnt;
  float               epsilon=1e-4; 
  float               dist_to_surf = 0.0;
  float               x,y,z;
  unsigned            rxn, done, cellnum;
  float               samp_dist = positions_buffer[launch_index].samp_dist;

  intersection_point  payload;
  
  float3 ray_direction  = make_float3(positions_buffer[launch_index].xhat, positions_buffer[launch_index].yhat, positions_buffer[launch_index].zhat);
  float3 ray_origin     = make_float3(positions_buffer[launch_index].x,    positions_buffer[launch_index].y,    positions_buffer[launch_index].z);
  optix::Ray ray        = optix::make_Ray( ray_origin, ray_direction, 0, epsilon, RT_DEFAULT_MAX );

  //rtPrintf("i=%u, (% 10.8E,% 10.8E,% 10.8E) (% 10.8E,% 10.8E,% 10.8E)\n",launch_index,positions_buffer[launch_index].x,positions_buffer[launch_index].y,positions_buffer[launch_index].z,positions_buffer[launch_index].xhat,positions_buffer[launch_index].yhat,positions_buffer[launch_index].zhat);

  // init payload
  payload.cont=1;
  payload.do_first_hit=1;
  for(cnt=0;cnt<10;cnt++){
    payload.hitbuff[cnt].cell = -1;
    payload.hitbuff[cnt].mat  = -1;
    payload.hitbuff[cnt].fiss = -1;
  }

  // first trace to find closest hit
  rtTrace(top_object, ray, payload);


   if (trace_type==1){   // transport trace type
      dist_to_surf = payload.surf_dist;
      if ( (dist_to_surf - 1.75*epsilon) >= samp_dist ){  // interaction is closer
         x = positions_buffer[launch_index].x + samp_dist*positions_buffer[launch_index].xhat;
         y = positions_buffer[launch_index].y + samp_dist*positions_buffer[launch_index].yhat;
         z = positions_buffer[launch_index].z + samp_dist*positions_buffer[launch_index].zhat;
         rxn = rxn_buffer[launch_index];
         done = 0;
         cellnum = cellnum_buffer[launch_index];
      } 
      else{ // surface is closer 
         if (payload.cell_first==outer_cell){ // first check if BC
            if(boundary_condition == 0){
              rxn  = 888;  //  set leak code
              done = 1;   // set done flag
              // move out of geometry to "interaction point"
              cellnum = payload.cell_first;
              x = positions_buffer[launch_index].x + samp_dist*positions_buffer[launch_index].xhat;
              y = positions_buffer[launch_index].y + samp_dist*positions_buffer[launch_index].yhat;
              z = positions_buffer[launch_index].z + samp_dist*positions_buffer[launch_index].zhat;
            }
            else if(boundary_condition == 1){
              rtPrintf("CRAPPPPPPPPP\n");
            }
            else{
              rtPrintf("DOUBLE CRAPPPPPPPPP\n");
            }
         }
         else{
          rxn = 999; // set resample code
          done = 0;
          // move to surface
          cellnum = payload.cell_first;
          x = positions_buffer[launch_index].x + dist_to_surf*positions_buffer[launch_index].xhat;
          y = positions_buffer[launch_index].y + dist_to_surf*positions_buffer[launch_index].yhat;
          z = positions_buffer[launch_index].z + dist_to_surf*positions_buffer[launch_index].zhat;
         }
      }
   //write positions to buffers
   rtPrintf("launch_index = %d rxn %u done %u cellnum %u xyz_o % 10.8E % 10.8E % 10.8E xyz % 10.8E % 10.8E % 10.8E samp %10.8E surf %10.8E \n",launch_index,rxn,done,cellnum,positions_buffer[launch_index].x,positions_buffer[launch_index].y,positions_buffer[launch_index].z,x,y,z,samp_dist,dist_to_surf);
   rxn_buffer[launch_index]  = rxn;
   done_buffer[launch_index] = done;
   cellnum_buffer[launch_index] = cellnum;
   positions_buffer[launch_index].x = x;
   positions_buffer[launch_index].y = y;
   positions_buffer[launch_index].z = z;
   }


   else if(trace_type==2 | trace_type==3){   // where am I? trace
      // check if bc
      if (payload.cell_first==outer_cell){
            payload.cont=0; 
            cellnum_buffer[launch_index]==outer_cell;
      }
      while(payload.cont){
         ray_origin = make_float3(payload.x,payload.y,payload.z);
         ray = optix::make_Ray( ray_origin, ray_direction, 0, epsilon, RT_DEFAULT_MAX );
         rtTrace(top_object, ray, payload);      
      }
      cellnum_buffer[launch_index] = payload.hitbuff[0].cell;
      // if number 4 requested, then write fissile flag to matnum instead of matnum
      //rtPrintf("cellnum,matnum,is_fissile = %d %d %d \n",payload.hitbuff[0].cell,payload.hitbuff[0].mat,payload.hitbuff[0].fiss);
      if(trace_type == 2){
        matnum_buffer[launch_index]=payload.hitbuff[0].mat;
      }
      else if(trace_type == 3){
        matnum_buffer[launch_index]=payload.hitbuff[0].fiss;
      }

   }


}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d)\n", code, launch_index);
  rtPrintExceptionDetails();
  //output_buffer[launch_index].x = -2;
  //output_buffer[launch_index].y = -2;
  //output_buffer[launch_index].z = -2;
}
