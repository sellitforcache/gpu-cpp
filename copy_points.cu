#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"

__global__ void copy_points_kernel( unsigned Nout, unsigned * Nvalid , unsigned current_index , unsigned * to_valid, source_point * positions_out , source_point * positions_in, float*E_out, float*E_in  ){

	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid >= Nvalid[0]){return;}

	unsigned index_in  = to_valid[tid];
	unsigned index_out = current_index + tid;
	if (index_out>=Nout){index_out=index_out-Nout;} //wrap to start
	//printf("index out = %u\n",index_out);

	// copy points
	positions_out[index_out].x    	= positions_in[index_in].x; 
	positions_out[index_out].y    	= positions_in[index_in].y; 
	positions_out[index_out].z    	= positions_in[index_in].z; 
	positions_out[index_out].xhat 	= positions_in[index_in].xhat; 
	positions_out[index_out].yhat 	= positions_in[index_in].yhat; 
	positions_out[index_out].zhat 	= positions_in[index_in].zhat;
	E_out[index_out] 				= E_in[index_in];

	//printf("good point %6.4E %6.4E %6.4E\n",positions_out[index_out].x,positions_out[index_out].y,positions_out[index_out].z);


}

void copy_points( unsigned NUM_THREADS,  unsigned Nout , unsigned * Nvalid,  unsigned current_index , unsigned * to_valid , source_point * positions_out , source_point * positions_in, float*E_out, float*E_in){

	unsigned blks = ( Nout + NUM_THREADS - 1 ) / NUM_THREADS;

	copy_points_kernel <<< blks, NUM_THREADS >>> (  Nout , Nvalid,  current_index , to_valid , positions_out , positions_in , E_out, E_in);
	hipDeviceSynchronize();

}

