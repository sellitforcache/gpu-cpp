#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"

__global__ void microscopic_kernel(unsigned N, unsigned n_isotopes, unsigned n_columns, unsigned* isonum, unsigned * index, float * main_E_grid, float * rn_bank, float * E, float * xs_data_MT , unsigned * xs_MT_numbers_total, unsigned * xs_MT_numbers,  float* xs_data_Q, unsigned * rxn, float* Q, unsigned* done){


	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid >= N){return;}
	if (done[tid]){return;}

	// load from array
	unsigned  	RNUM_PER_THREAD = 15;
	unsigned 	this_tope 		= isonum[tid];
	unsigned 	dex 			= index[tid];
	unsigned 	tope_beginning;
	unsigned 	tope_ending;
	unsigned 	this_dex;
	float 		this_E  		= E[tid];
	float 		rn1 			= rn_bank[tid*RNUM_PER_THREAD + 2];
	float 		cum_prob 		= 0.0;
	float 		this_Q 			= 0.0;
	unsigned 	k 				= 0;
	unsigned 	this_rxn 		= 999999999;

	if (this_tope == 0){  //first isotope
		tope_beginning = n_isotopes + 0;
		tope_ending    = n_isotopes + xs_MT_numbers_total[0]-1;
	}
	else{  //interior space
		tope_beginning = n_isotopes + xs_MT_numbers_total[this_tope-1];
		tope_ending    = n_isotopes + xs_MT_numbers_total[this_tope]-1;
	}

	//printf("tope,begin,end = %u %u %u\n",this_tope,tope_beginning,tope_ending);

	float xs_total = 0.0;
	float e0 = main_E_grid[dex];
	float e1 = main_E_grid[dex+1];
	float t0,t1;

	// compute the total microscopic cross section for this material
	// linearly interpolate, dex is the row number
	t0 			= xs_data_MT[n_columns* dex    + this_tope];     
	t1 			= xs_data_MT[n_columns*(dex+1) + this_tope];
	xs_total 	= (t1-t0)/(e1-e0)*(this_E-e0) + t0 ;    

	// determine the reaction for this isotope
	for(k=tope_beginning; k<tope_ending; k++){
		//lienarly interpolate
		t0 = xs_data_MT[n_columns* dex    + k];     
		t1 = xs_data_MT[n_columns*(dex+1) + k];
		cum_prob += ( (t1-t0)/(e1-e0)*(this_E-e0) + t0 ) / xs_total;
		if(rn1 <= cum_prob){
			// reactions happen in reaction k
			this_rxn = xs_MT_numbers[k];
			this_Q   = xs_data_Q[k];
			this_dex = k;
			break;
		}
	}

	if(this_rxn == 999999999){ // there is a gap in between the last MT and the total cross section, remap the rn to fit into the available data (effectively rescales the total cross section so everything adds up to it, if things aren't samples the first time around)
		//printf("REACTION NOT SAMPLED CORRECTLY! tope=%u E=%10.8E dex=%u rxn=%u cum_prob=%6.4E\n",this_tope, this_E, dex, this_rxn, cum_prob);
		rn1 = rn1 * cum_prob;
		cum_prob = 0.0;
		for(k=tope_beginning; k<tope_ending; k++){
			//lienarly interpolate
			t0 = xs_data_MT[n_columns* dex    + k];     
			t1 = xs_data_MT[n_columns*(dex+1) + k];
			cum_prob += ( (t1-t0)/(e1-e0)*(this_E-e0) + t0 ) / xs_total;
			if(rn1 <= cum_prob){
				// reactions happen in reaction k
				this_rxn = xs_MT_numbers[k];
				this_Q   = xs_data_Q[k];
				this_dex = k;
				break;
			}
		}
	}

	// write results out
	//if(this_rxn!=2){printf("this_rxn(%d,(1:3))=[%u,%u,%u];\n",tid+1,this_rxn,this_tope,k);}
	rxn[tid] = this_rxn;
	Q[tid] 	 = this_Q;
	//also write MT array index to dex instead of energy vector index
	index[tid] = this_dex;


}

void microscopic(unsigned blks, unsigned NUM_THREADS,  unsigned N, unsigned n_isotopes, unsigned n_columns, unsigned* isonum, unsigned * index, float * main_E_grid, float * rn_bank, float * E, float * xs_data_MT , unsigned * xs_MT_numbers_total, unsigned * xs_MT_numbers,  float* xs_data_Q, unsigned * rxn, float* Q, unsigned* done){

	microscopic_kernel <<< blks, NUM_THREADS >>> ( N, n_isotopes, n_columns, isonum, index, main_E_grid, rn_bank, E, xs_data_MT , xs_MT_numbers_total, xs_MT_numbers, xs_data_Q, rxn, Q, done);
	hipDeviceSynchronize();

}

