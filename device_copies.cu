#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"

void copy_to_device(void* dest,void* source ,unsigned bytes){

	hipMemcpy(dest,source,bytes,hipMemcpyHostToDevice);

}

void copy_from_device(void* dest,void* source ,unsigned bytes){

	hipMemcpy(dest,source,bytes,hipMemcpyDeviceToHost);

}