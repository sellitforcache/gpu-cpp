#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void reaction_edges_kernel( unsigned N, unsigned* edges, unsigned* rxn){

	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid >= N){return;}

	// the reaction vector has been sorted by this time.  data loads are expensive, computation is free, do as much as you can with 2 loads
	// need to find the lower AND upper bounds of the blocks.
	// array structure is:
	// 0  = done flag
	// 1  = lower bound for 2 block 
	// 2  = upper bound for 2 block
	// 3  = lower bound for 51/90 block 
	// 4  = upper bound for 51/90 block
	// 5  = lower bound for 91 block 
	// 6  = upper bound for 91 block
	// 7  = lower bound for 800 block 
	// 8  = upper bound for 800 block
	// 9  = lower bound for 811/845 block 
	// 10 = upper bound for 811/845 block

	unsigned rxn1, rxn2;
	int diff = 0; 

	// load data
	rxn1 = rxn[tid];
	if(tid < N-1){  //both elements if not last
		rxn2 = rxn[tid+1];
		diff = rxn2-rxn1;   //diff should be >0 since the list is sorted
		if(diff<0){printf("non-ascending value found in reaction list at index = %u (%u -> %u)\n!",tid,rxn1,rxn2);}
	}
	else{  //check last or only element, do not return

		if     (rxn1==2) 				{edges[2]  = tid+1;}
		else if(rxn1>=50 & rxn1<=90)	{edges[4]  = tid+1;}
		else if(rxn1==91)				{edges[6]  = tid+1;}
		else if(rxn1==800)				{edges[8]  = tid+1;}
		else if(rxn1>=811 & rxn1<=845)	{edges[10] = tid+1;}

	}

	// first (or only) element doesn't have a preceeding, write it in as the start of something, do not return
	if(tid==0){

		if     (rxn1==2) 				{edges[1] = 0;}
		else if(rxn1>=50 & rxn1<=90)	{edges[3] = 0;}
		else if(rxn1==91)				{edges[5] = 0;}
		else if(rxn1==800)				{edges[7] = 0;}
		else if(rxn1>=811 & rxn1<=845)	{edges[9] = 0;  edges[0]=1;}
		else if(rxn1>845)				{edges[0] = 1;}
		
	}

	// return if the same element, or if last/only element (diff will not be set and remain at 0)
	if(diff<0){return;}   

	// check edge
	if(rxn1<2 		& rxn2>=2)		{edges[1]  = tid+1;} //printf("setting starting edge of 2\n");}
	if(rxn1<=2 		& rxn2>2)		{edges[2]  = tid+1;} //printf("setting ending edge of 2\n");}
	if(rxn1<50 		& rxn2>=50)     {edges[3]  = tid+1;} //printf("setting starting edge of 50\n");}
	if(rxn1<=90 	& rxn2>90)		{edges[4]  = tid+1;} //printf("setting ending edge of 50\n");}
	if(rxn1<91 		& rxn2>=91)		{edges[5]  = tid+1;} //printf("setting starting edge of 91\n");}
	if(rxn1<=91 	& rxn2>91)		{edges[6]  = tid+1;} //printf("setting ending edge of 91\n");}
	if(rxn1<800 	& rxn2>=800)	{edges[7]  = tid+1;} //printf("setting starting edge of 800\n");}
	if(rxn1<=800 	& rxn2>800)		{edges[8]  = tid+1;} //printf("setting ending edge of 800\n");}
	if(rxn1<811 	& rxn2>=811)	{edges[9]  = tid+1;} //printf("setting starting edge of 811\n");}
	if(rxn1<=845  	& rxn2>845)		{edges[10] = tid+1;} //printf("setting ending edge of 811\n");}


}

void reaction_edges( unsigned NUM_THREADS,  unsigned N, unsigned* d_edges, unsigned* d_rxn){

	if(N<1){return;}
	unsigned blks = ( N + NUM_THREADS - 1 ) / NUM_THREADS;

	reaction_edges_kernel <<< blks, NUM_THREADS >>> ( N, d_edges, d_rxn);
	hipDeviceSynchronize();

}

