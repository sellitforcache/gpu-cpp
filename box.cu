#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtDeclareVariable(float3, mins, , );
rtDeclareVariable(float3, maxs, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int)
{
  float3 t0 = (mins - ray.origin)/ray.direction;
  float3 t1 = (maxs - ray.origin)/ray.direction;
  float3 near = fminf(t0, t1);
  float3 far = fmaxf(t0, t1);
  float tmin = fmaxf( near );
  float tmax = fminf( far );

  if(tmin <= tmax) {
    bool check_second = true;
    if( rtPotentialIntersection( tmin ) ) {
       if(rtReportIntersection(0))
         check_second = false;
    } 
    if(check_second) {
      if( rtPotentialIntersection( tmax ) ) {
        rtReportIntersection(0);
      }
    }
  }
}

RT_PROGRAM void bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(mins, maxs);
}
