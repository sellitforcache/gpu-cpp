#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtDeclareVariable(float3, mins, , );
rtDeclareVariable(float3, maxs, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int)
{
  float   tvec[8], tmin, ndD;
  float   ax, ay, az;
  float3  pvec[8];
  int     k;
  bool    report;

  // box/line region delimiters
  float x1 = maxs.y/sqrt(3.0);
  float x2 = 2*x1;
  
  // normal vectors
  float3 z_hat = make_float3( 0.0 , 0.0         , 1.0);
  float3 y_hat = make_float3( 0.0 , 1.0         , 0.0 );
  float3 r_hat = make_float3( 1.0 , 1/sqrt(3.0) , 0.0 );
  float3 l_hat = make_float3(-1.0 , 1/sqrt(3.0) , 0.0 );

  // points that define all planes
  float3 p4 = make_float3(  x2,  0           , mins.x );
  float3 p1 = make_float3(  x2,  0           , maxs.x );
  float3 p2 = make_float3( -x1,  maxs.y , maxs.x );
  float3 p3 = make_float3( -x1, -maxs.y , maxs.x );

  // find all plane intersections
  ndD=dot(z_hat,ray.direction);
  if (ndD!=0.0){
    tvec[0]=dot(z_hat,(p1-ray.origin))/ndD;
    tvec[1]=dot(z_hat,(p4-ray.origin))/ndD;
    pvec[0] = ray.origin+tvec[0]*ray.direction;
    pvec[1] = ray.origin+tvec[1]*ray.direction;
  }
  ndD=dot(y_hat,ray.direction);
  if (ndD!=0.0){
    tvec[2]=dot(  y_hat , ( p2 - ray.origin ) ) / ndD;
    tvec[3]=dot(  y_hat , ( p3 - ray.origin ) ) / ndD;
    pvec[2] = ray.origin+tvec[2]*ray.direction;
    pvec[3] = ray.origin+tvec[3]*ray.direction;
  }
  ndD=dot(l_hat,ray.direction);
  if (ndD!=0.0){
    tvec[4]=dot(  l_hat , ( p2 - ray.origin ) ) / ndD;
    tvec[5]=dot(  l_hat , ( p1 - ray.origin ) ) / ndD;
    pvec[4] = ray.origin+tvec[4]*ray.direction;
    pvec[5] = ray.origin+tvec[5]*ray.direction;
  }
  ndD=dot(r_hat,ray.direction);
  if (ndD!=0.0){
    tvec[6]=dot(  r_hat , ( p1 - ray.origin ) ) / ndD;
    tvec[7]=dot(  r_hat , ( p3 - ray.origin ) ) / ndD;
    pvec[6] = ray.origin+tvec[6]*ray.direction;
    pvec[7] = ray.origin+tvec[7]*ray.direction;
  }

  // get hits that are in-bounds (should only be 2, report one with smallest t)
    report=false;
    tmin=1.0/0.0;
    for (k=0;k<8;k++) {
        ax = fabsf(pvec[k].x);
        ay = fabsf(pvec[k].y);
        az = fabsf(pvec[k].z);
        // is in box region
        if (ax<=x1 && ay<=maxs.y) {
            if (az>=mins.x && az<=maxs.x) {
                if (tvec[k] >= 1e-8 && tvec[k] <= tmin) {
                    tmin=tvec[k];
                    report=true;
                }
            }
        }
        // is in line region
        else if (ax>x1 && ax<=x2 && (ay-(maxs.y*(2-ax/x1)))<=1e-6){
            if (az>=mins.x && az<=maxs.x) {
                if (tvec[k] >= 1e-8 && tvec[k] <= tmin) {
                    tmin=tvec[k];
                    report=true;
                }
            }
        }
    }

  // report t value of first intersection
  if(report) {
    if( rtPotentialIntersection( tmin ) ) {
       rtReportIntersection(0);
    }
  }

}

RT_PROGRAM void bounds (int, float result[6])
{
  result[0] = -2*maxs.y/sqrt(3.0);
  result[1] = -maxs.y;
  result[2] = mins.x;
  result[3] = 2*maxs.y/sqrt(3.0);
  result[4] = maxs.y;
  result[5] = maxs.x;
}
